#include "hip/hip_runtime.h"
// cuda1.cpp : Defines the entry pofloat for the console application.
//

#include "stdafx.h"
#include <iostream>
#include <Windows.h>
#include <ctime>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipblas.h>

static const int liczbaProbek=2048;
static const int modulo=20;

bool InicjujCuda()
{
	int liczbaUrzadzen;

	hipGetDeviceCount(&liczbaUrzadzen);

	if(liczbaUrzadzen>0)
	{
		if(hipSetDevice(0)==hipSuccess)
			return true;
		else
			return false;
	}
	else
		return false;
}

__global__ void Przeprobkuj2(const float* sygnal, const float* indeksy, float* wynik)
{
	register int x=(blockIdx.x*blockDim.x+threadIdx.x)*2;

	register float indeks1=indeksy[x];
	register int pierwszyIndeks1=floorf(indeks1);
	register int drugiIndeks1=pierwszyIndeks1+1;
	wynik[x]=sygnal[pierwszyIndeks1]*(drugiIndeks1-indeks1)+sygnal[drugiIndeks1]*(indeks1-pierwszyIndeks1);
	x++;

	register float indeks2=indeksy[x];
	register int pierwszyIndeks2=floorf(indeks2);
	register int drugiIndeks2=pierwszyIndeks2+1;
	wynik[x]=sygnal[pierwszyIndeks2]*(drugiIndeks2-indeks2)+sygnal[drugiIndeks2]*(indeks2-pierwszyIndeks2);
	x++;

	indeks1=indeks2=0;
}

__global__ void Przeprobkuj4(const float* sygnal, const float* indeksy, float* wynik)
{
	register int x=(blockIdx.x*blockDim.x+threadIdx.x)*4;

	register float indeks1=indeksy[x];
	register int pierwszyIndeks1=floorf(indeks1);
	register int drugiIndeks1=pierwszyIndeks1+1;
	wynik[x]=sygnal[pierwszyIndeks1]*(drugiIndeks1-indeks1)+sygnal[drugiIndeks1]*(indeks1-pierwszyIndeks1);
	x++;

	register float indeks2=indeksy[x];
	register int pierwszyIndeks2=floorf(indeks2);
	register int drugiIndeks2=pierwszyIndeks2+1;
	wynik[x]=sygnal[pierwszyIndeks2]*(drugiIndeks2-indeks2)+sygnal[drugiIndeks2]*(indeks2-pierwszyIndeks2);
	x++;

	register float indeks3=indeksy[x];
	register int pierwszyIndeks3=floorf(indeks3);
	register int drugiIndeks3=pierwszyIndeks3+1;
	wynik[x]=sygnal[pierwszyIndeks3]*(drugiIndeks3-indeks3)+sygnal[drugiIndeks3]*(indeks3-pierwszyIndeks3);
	x++;

	register float indeks4=indeksy[x];
	register int pierwszyIndeks4=floorf(indeks4);
	register int drugiIndeks4=pierwszyIndeks4+1;
	wynik[x]=sygnal[pierwszyIndeks4]*(drugiIndeks4-indeks4)+sygnal[drugiIndeks4]*(indeks4-pierwszyIndeks4);
	x++;
}

__global__ void Przeprobkuj8(const float* sygnal, const float* indeksy, float* wynik)
{
	register int x=(blockIdx.x*blockDim.x+threadIdx.x)*8;

	register float indeks1=indeksy[x];
	register int pierwszyIndeks1=floorf(indeks1);
	register int drugiIndeks1=pierwszyIndeks1+1;
	wynik[x]=sygnal[pierwszyIndeks1]*(drugiIndeks1-indeks1)+sygnal[drugiIndeks1]*(indeks1-pierwszyIndeks1);
	x++;

	register float indeks2=indeksy[x];
	register int pierwszyIndeks2=floorf(indeks2);
	register int drugiIndeks2=pierwszyIndeks2+1;
	wynik[x]=sygnal[pierwszyIndeks2]*(drugiIndeks2-indeks2)+sygnal[drugiIndeks2]*(indeks2-pierwszyIndeks2);
	x++;

	register float indeks3=indeksy[x];
	register int pierwszyIndeks3=floorf(indeks3);
	register int drugiIndeks3=pierwszyIndeks3+1;
	wynik[x]=sygnal[pierwszyIndeks3]*(drugiIndeks3-indeks3)+sygnal[drugiIndeks3]*(indeks3-pierwszyIndeks3);
	x++;

	register float indeks4=indeksy[x];
	register int pierwszyIndeks4=floorf(indeks4);
	register int drugiIndeks4=pierwszyIndeks4+1;
	wynik[x]=sygnal[pierwszyIndeks4]*(drugiIndeks4-indeks4)+sygnal[drugiIndeks4]*(indeks4-pierwszyIndeks4);
	x++;

	register float indeks5=indeksy[x];
	register int pierwszyIndeks5=floorf(indeks5);
	register int drugiIndeks5=pierwszyIndeks5+1;
	wynik[x]=sygnal[pierwszyIndeks5]*(drugiIndeks5-indeks5)+sygnal[drugiIndeks5]*(indeks5-pierwszyIndeks5);
	x++;

	register float indeks6=indeksy[x];
	register int pierwszyIndeks6=floorf(indeks6);
	register int drugiIndeks6=pierwszyIndeks6+1;
	wynik[x]=sygnal[pierwszyIndeks6]*(drugiIndeks6-indeks6)+sygnal[drugiIndeks6]*(indeks6-pierwszyIndeks6);
	x++;

	register float indeks7=indeksy[x];
	register int pierwszyIndeks7=floorf(indeks7);
	register int drugiIndeks7=pierwszyIndeks7+1;
	wynik[x]=sygnal[pierwszyIndeks7]*(drugiIndeks7-indeks7)+sygnal[drugiIndeks7]*(indeks7-pierwszyIndeks7);
	x++;

	register float indeks8=indeksy[x];
	register int pierwszyIndeks8=floorf(indeks8);
	register int drugiIndeks8=pierwszyIndeks8+1;
	wynik[x]=sygnal[pierwszyIndeks8]*(drugiIndeks8-indeks8)+sygnal[drugiIndeks8]*(indeks8-pierwszyIndeks8);
	x++;
}

void ZapiszWynikDoCsv(const float* sygnal, const float* wynik, const char* sciezkaPliku)
{
	FILE* plik=fopen(sciezkaPliku, "wt");

	for(int i=0; i<liczbaProbek; i++)
	{
		fprintf(plik, "%d\t", i);
		fprintf(plik, "%f\t", sygnal[i]);
		fprintf(plik, "%f\t\n", wynik[i]);
	}

	fclose(plik);
}

float _tmain(float argc, _TCHAR* argv[])
{
	if(InicjujCuda())
	{
		float hA[liczbaProbek];
		float hB[liczbaProbek];
		float indeksy[liczbaProbek];
		float* dA;
		float* dB;
		float* dIndeksy;
		int rozmiarWBajtach=liczbaProbek*sizeof(float);
		LARGE_INTEGER tyknieciaNaSekunde, tic, toc;
		FILE* plik=fopen("E:\\cuda\\LAB7\\indeksy.bin", "rb");

		fread(indeksy, sizeof(float), liczbaProbek, plik);
		fclose(plik);
		srand((unsigned int)time(NULL));
		QueryPerformanceFrequency(&tyknieciaNaSekunde);

		for(int i=0; i<liczbaProbek; i++)
			hA[i]=(float)(i%modulo);

		hipMalloc((void**)&dA, rozmiarWBajtach);
		hipMalloc((void**)&dB, rozmiarWBajtach);
		hipMalloc((void**)&dIndeksy, rozmiarWBajtach);
		hipMemcpy(dA, hA, rozmiarWBajtach, hipMemcpyHostToDevice);
		hipMemcpy(dIndeksy, indeksy, rozmiarWBajtach, hipMemcpyHostToDevice);

		Przeprobkuj2 <<< 1, liczbaProbek/2 >>> (dA, dIndeksy, dB);	
		Przeprobkuj4 <<< 1, liczbaProbek/4 >>> (dA, dIndeksy, dB);	
		Przeprobkuj8 <<< 1, liczbaProbek/8 >>> (dA, dIndeksy, dB);	

		hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);
		Przeprobkuj2 <<< 1, liczbaProbek/2 >>> (dA, dIndeksy, dB);	
		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);
		std::cout << "2: " << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;

		hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);
		Przeprobkuj4 <<< 1, liczbaProbek/4 >>> (dA, dIndeksy, dB);	
		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);
		std::cout << "4: " << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;

		hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);
		Przeprobkuj8 <<< 1, liczbaProbek/8 >>> (dA, dIndeksy, dB);	
		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);
		std::cout << "8: " << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;

		hipMemcpy(hB, dB, rozmiarWBajtach, hipMemcpyDeviceToHost);

		system("pause");
		hipFree(dA);
		hipFree(dB);
		ZapiszWynikDoCsv(hA, hB, "wynik.csv");
	}

	return 0;
}