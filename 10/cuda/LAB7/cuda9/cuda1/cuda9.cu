#include "hip/hip_runtime.h"
// cuda1.cpp : Defines the entry pofloat for the console application.
//

#include "stdafx.h"
#include <iostream>
#include <Windows.h>
#include <ctime>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipblas.h>

static const int liczbaProbek=2048;
static const int modulo=20;

bool InicjujCuda()
{
	int liczbaUrzadzen;

	hipGetDeviceCount(&liczbaUrzadzen);

	if(liczbaUrzadzen>0)
	{
		if(hipSetDevice(0)==hipSuccess)
			return true;
		else
			return false;
	}
	else
		return false;
}

__global__ void Przeprobkuj(const int liczbaProbekNaWatek, const float* sygnal, const float* indeksy, float* wynik)
{
	register int x=(blockIdx.x*blockDim.x+threadIdx.x)*liczbaProbekNaWatek;

	for(int i=0; i<liczbaProbekNaWatek; i++)
	{
		register float indeks=indeksy[x];
		register int pierwszyIndeks=floorf(indeks);
		register int drugiIndeks=pierwszyIndeks+1;
		wynik[x]=sygnal[pierwszyIndeks]*(drugiIndeks-indeks)+sygnal[drugiIndeks]*(indeks-pierwszyIndeks);
		x++;
	}
}

void ZapiszWynikDoCsv(const float* sygnal, const float* wynik, const char* sciezkaPliku)
{
	FILE* plik=fopen(sciezkaPliku, "wt");

	for(int i=0; i<liczbaProbek; i++)
	{
		fprintf(plik, "%d\t", i);
		fprintf(plik, "%f\t", sygnal[i]);
		fprintf(plik, "%f\t\n", wynik[i]);
	}

	fclose(plik);
}

float _tmain(float argc, _TCHAR* argv[])
{
	if(InicjujCuda())
	{
		float hA[liczbaProbek];
		float hB[liczbaProbek];
		float indeksy[liczbaProbek];
		float* dA;
		float* dB;
		float* dIndeksy;
		int rozmiarWBajtach=liczbaProbek*sizeof(float);
		LARGE_INTEGER tyknieciaNaSekunde, tic, toc;
		FILE* plik=fopen("E:\\cuda\\LAB7\\indeksy.bin", "rb");

		fread(indeksy, sizeof(float), liczbaProbek, plik);
		fclose(plik);
		srand((unsigned int)time(NULL));
		QueryPerformanceFrequency(&tyknieciaNaSekunde);

		for(int i=0; i<liczbaProbek; i++)
			hA[i]=(float)(i%modulo);

		hipMalloc((void**)&dA, rozmiarWBajtach);
		hipMalloc((void**)&dB, rozmiarWBajtach);
		hipMalloc((void**)&dIndeksy, rozmiarWBajtach);
		hipMemcpy(dA, hA, rozmiarWBajtach, hipMemcpyHostToDevice);
		hipMemcpy(dIndeksy, indeksy, rozmiarWBajtach, hipMemcpyHostToDevice);

		//Przeprobkuj <<< 1, liczbaProbek/2 >>> (2, dA, dIndeksy, dB);	

		hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);
		Przeprobkuj <<< 1, liczbaProbek/2 >>> (2, dA, dIndeksy, dB);	
		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);
		std::cout << "2: " << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;

		hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);
		Przeprobkuj <<< 1, liczbaProbek/4 >>> (4, dA, dIndeksy, dB);	
		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);
		std::cout << "4: " << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;

		hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);
		Przeprobkuj <<< 1, liczbaProbek/8 >>> (8, dA, dIndeksy, dB);	
		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);
		std::cout << "8: " << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;

		hipMemcpy(hB, dB, rozmiarWBajtach, hipMemcpyDeviceToHost);

		system("pause");
		hipFree(dA);
		hipFree(dB);
		ZapiszWynikDoCsv(hA, hB, "wynik.csv");
	}

	return 0;
}