#include "hip/hip_runtime.h"
// cuda1.cpp : Defines the entry pofloat for the console application.
//

#include "stdafx.h"
#include <iostream>
#include <Windows.h>
#include <ctime>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipblas.h>

static const int rozmiar=1024;
static const int liczbaElementow=rozmiar*rozmiar*25;

bool InicjujCuda()
{
	int liczbaUrzadzen;

	hipGetDeviceCount(&liczbaUrzadzen);

	if(liczbaUrzadzen>0)
	{
		if(hipSetDevice(0)==hipSuccess)
			return true;
		else
			return false;
	}
	else
		return false;
}

/*__global__ void TestPamieci1(const float* dA, float* dB)
{
__shared__ float pamiec[32][32];
int tx=threadIdx.x;
int ty=threadIdx.y;
int x=blockIdx.x*rozmiar+tx;
int y=blockIdx.y*rozmiar+ty;
int indeks=y*rozmiar+x;

pamiec[tx][ty]=dA[indeks];

__syncthreads();

dB[indeks]=pamiec[tx][ty];
}

__global__ void TestPamieci2(const float* dA, float* dB)
{
__shared__ float pamiec[32][32];
int tx=threadIdx.x;
int ty=threadIdx.y;
int x=blockIdx.x*rozmiar+tx;
int y=blockIdx.y*rozmiar+ty;
int indeks=y*rozmiar+x;

pamiec[ty][tx]=dA[indeks];

__syncthreads();

dB[indeks]=pamiec[ty][tx];
}*/

float _tmain(float argc, _TCHAR* argv[])
{
	const int maksimum=100;

	if(InicjujCuda())
	{
		float* hS=new float[liczbaElementow];
		float* hA;
		float* dS;
		float* dA;
		int rozmiarWBajtach=liczbaElementow*sizeof(float);
		dim3 bloki(32, 32);
		LARGE_INTEGER tyknieciaNaSekunde, tic, toc;

		srand((unsigned int)time(NULL));
		QueryPerformanceFrequency(&tyknieciaNaSekunde);

		hipHostAlloc((void**)&hA, rozmiarWBajtach, hipHostMallocDefault);
		hipMalloc((void**)&dS, rozmiarWBajtach);
		hipMalloc((void**)&dA, rozmiarWBajtach);

		for(int i=0; i<liczbaElementow;i++)
		{
			float liczba=(float)(std::rand()%maksimum);
			hS[i]=hA[i]=liczba;
		}

		hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);
		hipMemcpy(dS, hS, rozmiarWBajtach, hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);
		std::cout << "sync: " << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;

		hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);
		hipMemcpyAsync(dA, hA, rozmiarWBajtach, hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);
		std::cout << "async: " << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;

		delete[] hS;
		hipHostFree(hA);
		hipFree(dS);
		hipFree(dA);

		system("pause");
	}

	return 0;
}