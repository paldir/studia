#include "hip/hip_runtime.h"
// cuda1.cpp : Defines the entry pofloat for the console application.
//

#include "stdafx.h"
#include <iostream>
#include <Windows.h>
#include <ctime>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipblas.h>

static const int liczbaBlokow=10000;
static const int liczbaWatkowWBloku=512;
static const int liczbaElementow=liczbaBlokow*liczbaWatkowWBloku*4;

bool InicjujCuda()
{
	int liczbaUrzadzen;

	hipGetDeviceCount(&liczbaUrzadzen);

	if(liczbaUrzadzen>0)
	{
		if(hipSetDevice(0)==hipSuccess)
			return true;
		else
			return false;
	}
	else
		return false;
}

__global__ void kernel(float* d_in, float* d_out)
{
	register int index = blockIdx.x * blockDim.x + threadIdx.x;
	// unikatowy identyfikator w�tku
	register float a = d_in[4*index];
	register float b = d_in[4*index+1];
	register float c = d_in[4*index+2];
	register float d = d_in[4*index+3];
	// wczytanie danych z pami�ci karty graficznej
	a =b*c; // opcjonalne
	b =c+d; // operacje
	c =a*b; // matematyczne
	d =a+c; //
	d_out[4*index] = a;
	d_out[4*index+1] = b;
	d_out[4*index+2] = c;
	d_out[4*index+3] = d;
	// zapis danych do pami�ci karty graficznej
}

void launch_kernel(float* d_in, float* d_out)
{
	int nBlocks = 10000;
	// 10 000 blok�w w�tk�w
	int threadsPerBlock = 512;
	// po 512 w�tk�w ka�dy
	kernel<<<nBlocks, threadsPerBlock>>>(d_in,d_out);
	// wywo�anie funkcji j�dra za pomoc� ponad 5 mln w�tk�w
	hipDeviceSynchronize();
	//oczekiwanie na zako�czenie pracy w�tk�w
}

float _tmain(float argc, _TCHAR* argv[])
{
	const int maksimum=100;

	if(InicjujCuda())
	{
		float* hS=new float[liczbaElementow];
		float* dS;
		float* dT;
		int rozmiarWBajtach=liczbaElementow*sizeof(float);
		dim3 bloki(32, 32);
		LARGE_INTEGER tyknieciaNaSekunde, tic, toc;

		srand((unsigned int)time(NULL));
		QueryPerformanceFrequency(&tyknieciaNaSekunde);
		hipMalloc((void**)&dS, rozmiarWBajtach);
		hipMalloc((void**)&dT, rozmiarWBajtach);

		for(int i=0; i<liczbaElementow;i++)
		{
			float liczba=(float)(std::rand()%maksimum);
			hS[i]=liczba;
		}

		hipMemcpy(dS, hS, rozmiarWBajtach, hipMemcpyHostToDevice);

		hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);
		launch_kernel(dS, dT);
		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);
		std::cout << "czas: " << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;

		delete[] hS;
		hipFree(dS);
		hipFree(dT);

		system("pause");
	}

	return 0;
}