// cuda1.cpp : Defines the entry point for the console application.
//

#include "stdafx.h"
#include <iostream>
#include <hip/hip_runtime.h>

int _tmain(int argc, _TCHAR* argv[])
{
	int liczbaUrzadzen=0;
	hipDeviceProp_t wlasciwosciUrzadzenia;

	hipGetDeviceCount(&liczbaUrzadzen);

	std::cout << "Liczba urzadzen: " << liczbaUrzadzen << std::endl;

	for(int i=0; i<liczbaUrzadzen;i++)
	{
		int* maxThreadsDim=wlasciwosciUrzadzenia.maxThreadsDim;
		int* maxGridSize=wlasciwosciUrzadzenia.maxGridSize;
		
		hipGetDeviceProperties(&wlasciwosciUrzadzenia, i);

		std::cout << "Nazwa: " << wlasciwosciUrzadzenia.name << std::endl;
		std::cout << "Calkowita ilosc pamieci: " << wlasciwosciUrzadzenia.totalGlobalMem << std::endl;
		std::cout << "Pamiec dzielona na blok: " << wlasciwosciUrzadzenia.sharedMemPerBlock << std::endl;
		std::cout << "Rejestry na blok: " << wlasciwosciUrzadzenia.regsPerBlock << std::endl;
		std::cout << "Rozmiar osnowy: " << wlasciwosciUrzadzenia.warpSize << std::endl;
		std::cout << "Maksymalna liczba watkow na blok: " << wlasciwosciUrzadzenia.maxThreadsPerBlock << std::endl;
		std::cout << "Maksymalny rozmiar kazdego wymiaru w bloku: " << maxThreadsDim[0] << " " << maxThreadsDim[1] << " " << maxThreadsDim[2] << std::endl;
		std::cout << "Maksymalny rozmiar kazdego wymiaru w siatce: " << maxGridSize[0] << " " << maxGridSize[1] << " " << maxGridSize[2] << std::endl;
		std::cout << "Taktowanie zegara: " << wlasciwosciUrzadzenia.clockRate << std::endl;
	}

	if(liczbaUrzadzen>0)
		hipSetDevice(0);

	getchar();

	return 0;
}