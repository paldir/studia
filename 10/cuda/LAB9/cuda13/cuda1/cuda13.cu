#include "hip/hip_runtime.h"
// cuda1.cpp : Defines the entry pofloat for the console application.
//

#include "stdafx.h"
#include <iostream>
#include <Windows.h>
#include <ctime>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipblas.h>

static const int rozmiar=1024;
static const int liczbaElementow=rozmiar*rozmiar*25;
static const int liczbaPodzialow=4;
static const int liczbaElementowWBuforze=liczbaElementow/liczbaPodzialow;

bool InicjujCuda()
{
	int liczbaUrzadzen;

	hipGetDeviceCount(&liczbaUrzadzen);

	if(liczbaUrzadzen>0)
	{
		if(hipSetDevice(0)==hipSuccess)
			return true;
		else
			return false;
	}
	else
		return false;
}

__global__ void DodajWektory(const float* dA, const float* dB, float* dC)
{
	int x=blockIdx.x*blockDim.x+threadIdx.x;
	int a=dA[x];
	int b=dB[x];
	int c=a+b;
	dC[x]=c;
}

float _tmain(float argc, _TCHAR* argv[])
{
	const int maksimum=100;

	if(InicjujCuda())
	{
		float* hA;
		float* hB;
		float* hC1;
		float* hC2;
		float* hC3;
		float* hC4;
		float* dA;
		float* dB;
		float* dC;
		float* dA2;
		float* dB2;
		float* dC2;
		int rozmiarWBajtach=liczbaElementow*sizeof(float);
		int rozmiarBufora=liczbaElementowWBuforze*sizeof(float);
		hipEvent_t tic, toc;
		float czas;
		hipStream_t strumien;
		hipStream_t strumien2;

		srand((unsigned int)time(NULL));
		hipEventCreate(&tic);
		hipEventCreate(&toc);
		hipStreamCreate(&strumien);
		hipStreamCreate(&strumien2);
		hipHostAlloc((void**)&hA, rozmiarWBajtach, hipHostMallocDefault);
		hipHostAlloc((void**)&hB, rozmiarWBajtach, hipHostMallocDefault);
		hC1=new float[liczbaElementow];
		hipHostAlloc((void**)&hC2, rozmiarWBajtach, hipHostMallocDefault);
		hipHostAlloc((void**)&hC3, rozmiarWBajtach, hipHostMallocDefault);
		hipHostAlloc((void**)&hC4, rozmiarWBajtach, hipHostMallocDefault);
		hipMalloc((void**)&dA, rozmiarBufora);
		hipMalloc((void**)&dB, rozmiarBufora);
		hipMalloc((void**)&dC, rozmiarBufora);
		hipMalloc((void**)&dA2, rozmiarBufora);
		hipMalloc((void**)&dB2, rozmiarBufora);
		hipMalloc((void**)&dC2, rozmiarBufora);

		for(int i=0; i<liczbaElementow;i++)
		{
			hA[i]=(float)(std::rand()%maksimum);
			hB[i]=(float)(std::rand()%maksimum);
		}

		//---------------------------------------------------------------------------------

		hipDeviceSynchronize();
		hipEventRecord(tic);

		for(int i=0; i<liczbaElementow; i+=liczbaElementowWBuforze)
		{
			hipMemcpy(dA, hA+i, rozmiarBufora, hipMemcpyHostToDevice);
			hipMemcpy(dB, hB+i, rozmiarBufora, hipMemcpyHostToDevice);

			DodajWektory <<< liczbaElementowWBuforze/256, 256 >>> (dA, dB, dC);

			hipMemcpy(hC1+i, dC, rozmiarBufora, hipMemcpyDeviceToHost);
		}

		hipDeviceSynchronize();
		hipEventRecord(toc);
		hipEventSynchronize(toc);
		hipEventElapsedTime(&czas, tic, toc);

		std::cout << "1: " << czas << std::endl;

		//---------------------------------------------------------------------------------

		hipDeviceSynchronize();
		hipEventRecord(tic, strumien);

		for(int i=0; i<liczbaElementow; i+=liczbaElementowWBuforze)
		{
			hipMemcpyAsync(dA, hA+i, rozmiarBufora, hipMemcpyHostToDevice, strumien);
			hipMemcpyAsync(dB, hB+i, rozmiarBufora, hipMemcpyHostToDevice, strumien);

			DodajWektory <<< liczbaElementowWBuforze/256, 256, 0, strumien >>> (dA, dB, dC);

			hipMemcpyAsync(hC2+i, dC, rozmiarBufora, hipMemcpyDeviceToHost, strumien);
		}

		hipStreamSynchronize(strumien);
		hipEventRecord(toc, strumien);
		hipEventSynchronize(toc);
		hipEventElapsedTime(&czas, tic, toc);

		std::cout << "2: " << czas << std::endl;

		//---------------------------------------------------------------------------------

		hipDeviceSynchronize();
		hipEventRecord(tic, strumien);

		for(int i=0; i<liczbaElementow; i+=liczbaElementowWBuforze*2)
		{
			hipMemcpyAsync(dA, hA+i, rozmiarBufora, hipMemcpyHostToDevice, strumien);
			hipMemcpyAsync(dB, hB+i, rozmiarBufora, hipMemcpyHostToDevice, strumien);

			DodajWektory <<< liczbaElementowWBuforze/256, 256, 0, strumien >>> (dA, dB, dC);

			hipMemcpyAsync(hC3+i, dC, rozmiarBufora, hipMemcpyDeviceToHost, strumien);


			hipMemcpyAsync(dA, hA+i+liczbaElementowWBuforze, rozmiarBufora, hipMemcpyHostToDevice, strumien2);
			hipMemcpyAsync(dB, hB+i+liczbaElementowWBuforze, rozmiarBufora, hipMemcpyHostToDevice, strumien2);

			DodajWektory <<< liczbaElementowWBuforze/256, 256, 0, strumien2 >>> (dA, dB, dC);

			hipMemcpyAsync(hC3+i+liczbaElementowWBuforze, dC, rozmiarBufora, hipMemcpyDeviceToHost, strumien2);
		}

		hipStreamSynchronize(strumien2);
		hipEventRecord(toc, strumien2);
		hipEventSynchronize(toc);
		hipEventElapsedTime(&czas, tic, toc);

		std::cout << "3: " << czas << std::endl;

		//---------------------------------------------------------------------------------

		hipDeviceSynchronize();
		hipEventRecord(tic, strumien);

		for(int i=0; i<liczbaElementow; i+=liczbaElementowWBuforze*2)
		{
			hipMemcpyAsync(dA, hA+i, rozmiarBufora, hipMemcpyHostToDevice, strumien);
			hipMemcpyAsync(dA2, hA+i+liczbaElementowWBuforze, rozmiarBufora, hipMemcpyHostToDevice, strumien2);
			hipMemcpyAsync(dB, hB+i, rozmiarBufora, hipMemcpyHostToDevice, strumien);
			hipMemcpyAsync(dB2, hB+i+liczbaElementowWBuforze, rozmiarBufora, hipMemcpyHostToDevice, strumien2);

			DodajWektory <<< liczbaElementowWBuforze/256, 256, 0, strumien >>> (dA, dB, dC);
			DodajWektory <<< liczbaElementowWBuforze/256, 256, 0, strumien2 >>> (dA2, dB2, dC2);

			hipMemcpyAsync(hC4+i, dC, rozmiarBufora, hipMemcpyDeviceToHost, strumien);
			hipMemcpyAsync(hC4+i+liczbaElementowWBuforze, dC2, rozmiarBufora, hipMemcpyDeviceToHost, strumien2);
		}

		hipStreamSynchronize(strumien2);
		hipEventRecord(toc, strumien2);
		hipEventSynchronize(toc);
		hipEventElapsedTime(&czas, tic, toc);

		std::cout << "4: " << czas << std::endl;

		//---------------------------------------------------------------------------------

		for(int i=0; i<liczbaElementow; i++)
		{
			if(hC1[i]!=hA[i]+hB[i])
			{
				std:: cout << "Blad w pierwszej metodzie.";

				break;
			}
		}

		for(int i=0; i<liczbaElementow; i++)
		{
			if(hC2[i]!=hA[i]+hB[i])
			{
				std:: cout << "Blad w drugiej metodzie.";

				break;
			}
		}	

		for(int i=0; i<liczbaElementow; i++)
		{
			if(hC3[i]!=hA[i]+hB[i])
			{
				std:: cout << "Blad w trzeciej metodzie.";

				break;
			}
		}

		for(int i=0; i<liczbaElementow; i++)
		{
			if(hC4[i]!=hA[i]+hB[i])
			{
				std:: cout << "Blad w czwartej metodzie.";

				break;
			}
		}

		hipHostFree(hA);
		hipHostFree(hB);
		delete[] hC1;
		hipHostFree(hC2);
		hipHostFree(hC3);
		hipHostFree(hC4);
		hipFree(dA);
		hipFree(dB);
		hipFree(dC);
		hipFree(dA2);
		hipFree(dB2);
		hipFree(dC2);
		hipStreamDestroy(strumien);
		hipStreamDestroy(strumien2);

		system("pause");
	}

	return 0;
}