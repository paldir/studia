#include "hip/hip_runtime.h"
// cuda1.cpp : Defines the entry point for the console application.
//

#include "stdafx.h"
#include <iostream>
#include <hip/hip_runtime.h>

bool InicjujCuda()
{
	int liczbaUrzadzen;

	hipGetDeviceCount(&liczbaUrzadzen);

	if(liczbaUrzadzen>0)
	{
		hipSetDevice(0);

		return true;
	}
	else
		return false;
}

__global__ void Dodaj(float* dA, float* dB, float* dC)
{
	int idWatku=threadIdx.x;

	dC[idWatku]=dA[idWatku]+dB[idWatku];
}

int _tmain(int argc, _TCHAR* argv[])
{
	if(InicjujCuda())
	{
		const int rozmiarWektorow=32;
		float hA[rozmiarWektorow];
		float hB[rozmiarWektorow];
		float hC[rozmiarWektorow];
		float hCLokalne[rozmiarWektorow];
		float* dA;
		float* dB;
		float* dC;
		int rozmiarTablicyWBajtach=rozmiarWektorow*sizeof(float);

		for(int i=0; i<rozmiarWektorow;i++)
		{
			hA[i]=hB[i]=i+1.0f;
			hCLokalne[i]=hA[i]+hB[i];
		}

		hipMalloc((void**)&dA, rozmiarTablicyWBajtach);
		hipMalloc((void**)&dB, rozmiarTablicyWBajtach);
		hipMalloc((void**)&dC, rozmiarTablicyWBajtach);
		hipMemcpy(dA, hA, rozmiarTablicyWBajtach, hipMemcpyHostToDevice);
		hipMemcpy(dB, hB, rozmiarTablicyWBajtach, hipMemcpyHostToDevice);

		std::cout << "Przed sumowaniem na GPU: " << std::endl;

		for(int i=0; i<rozmiarWektorow;i++)
			std::cout << hA[i] << "\t" << hB[i] << "\t" << hC[i] << std::endl;

		system("pause");

		Dodaj <<< 1, 32 >>> (dA, dB, dC);

		hipMemcpy(hC, dC, rozmiarTablicyWBajtach, hipMemcpyDeviceToHost);

		std::cout << "Po sumowaniu na GPU: " << std::endl;
		
		for(int i=0; i<rozmiarWektorow;i++)
			std::cout << hA[i] << "\t" << hB[i] << "\t" << hC[i] << "\t" << hCLokalne[i] << std::endl;
		
		system("pause");
	}

	return 0;
}