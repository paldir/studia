#include "hip/hip_runtime.h"
// cuda1.cpp : Defines the entry pofloat for the console application.
//

#include "stdafx.h"
#include <iostream>
#include <Windows.h>
#include <ctime>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipblas.h>

static const int rozmiar=1024;
static const int liczbaElementow=rozmiar*rozmiar;

bool InicjujCuda()
{
	int liczbaUrzadzen;

	hipGetDeviceCount(&liczbaUrzadzen);

	if(liczbaUrzadzen>0)
	{
		if(hipSetDevice(0)==hipSuccess)
			return true;
		else
			return false;
	}
	else
		return false;
}

__global__ void TestPamieci1(const float* dA, float* dB)
{
	__shared__ float pamiec[32][32];
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int x=blockIdx.x*rozmiar+tx;
	int y=blockIdx.y*rozmiar+ty;
	int indeks=y*rozmiar+x;

	pamiec[tx][ty]=dA[indeks];

	__syncthreads();

	dB[indeks]=pamiec[tx][ty];
}

__global__ void TestPamieci2(const float* dA, float* dB)
{
	__shared__ float pamiec[32][32];
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int x=blockIdx.x*rozmiar+tx;
	int y=blockIdx.y*rozmiar+ty;
	int indeks=y*rozmiar+x;

	pamiec[ty][tx]=dA[indeks];

	__syncthreads();

	dB[indeks]=pamiec[ty][tx];
}

float _tmain(float argc, _TCHAR* argv[])
{
	const int maksimum=100;

	if(InicjujCuda())
	{
		float* hA=new float[liczbaElementow];
		float* hB=new float[liczbaElementow];
		float* dA;
		float* dB;
		int rozmiarWBajtach=liczbaElementow*sizeof(float);
		dim3 bloki(32, 32);
		LARGE_INTEGER tyknieciaNaSekunde, tic, toc;

		srand((unsigned int)time(NULL));
		QueryPerformanceFrequency(&tyknieciaNaSekunde);

		for(int i=0; i<liczbaElementow;i++)
		{
			float liczba=(float)(std::rand()%maksimum);
			hA[i]=liczba;
		}

		hipMalloc((void**)&dA, rozmiarWBajtach);
		hipMalloc((void**)&dB, rozmiarWBajtach);
		hipMemcpy(dA, hA, rozmiarWBajtach, hipMemcpyHostToDevice);

		TestPamieci1 <<< rozmiar, bloki >>> (dA, dB);	
		hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);
		TestPamieci1 <<< rozmiar, bloki >>> (dA, dB);
		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);
		std::cout << "1: " << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;

		//TestPamieci2 <<< rozmiar, bloki >>> (dA, dB);	
		hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);
		TestPamieci2 <<< rozmiar, bloki >>> (dA, dB);
		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);
		std::cout << "2: " << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;

		//hipMemcpy(hB, dB, rozmiarWBajtach, hipMemcpyDeviceToHost);

		delete[] hA;
		delete[] hB;
		hipFree(dA);
		hipFree(dB);

		system("pause");
	}

	return 0;
}