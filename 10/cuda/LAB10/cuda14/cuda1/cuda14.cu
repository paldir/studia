#include "hip/hip_runtime.h"
// cuda1.cpp : Defines the entry pofloat for the console application.
//

#include "stdafx.h"
#include <iostream>
#include <Windows.h>
#include <ctime>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>

static const int N=1024;
static const int modulo=20;

bool InicjujCuda()
{
	int liczbaUrzadzen;

	hipGetDeviceCount(&liczbaUrzadzen);

	if(liczbaUrzadzen>0)
	{
		if(hipSetDevice(0)==hipSuccess)
			return true;
		else
			return false;
	}
	else
		return false;
}

__global__ void Przeprobkuj4(const hipfftReal* sygnal, const hipfftReal* indeksy, hipfftReal* wynik)
{
	register int x=(blockIdx.x*blockDim.x+threadIdx.x);

	register hipfftReal indeks1=indeksy[x]*4;
	register int pierwszyIndeks1=floorf(indeks1);
	register int drugiIndeks1=pierwszyIndeks1+1;
	wynik[x]=sygnal[pierwszyIndeks1]*(drugiIndeks1-indeks1)+sygnal[drugiIndeks1]*(indeks1-pierwszyIndeks1);

	/*register hipfftReal indeks2=indeksy[x];
	register int pierwszyIndeks2=floorf(indeks2);
	register int drugiIndeks2=pierwszyIndeks2+1;
	wynik[x]=sygnal[pierwszyIndeks2]*(drugiIndeks2-indeks2)+sygnal[drugiIndeks2]*(indeks2-pierwszyIndeks2);
	x++;

	register hipfftReal indeks3=indeksy[x];
	register int pierwszyIndeks3=floorf(indeks3);
	register int drugiIndeks3=pierwszyIndeks3+1;
	wynik[x]=sygnal[pierwszyIndeks3]*(drugiIndeks3-indeks3)+sygnal[drugiIndeks3]*(indeks3-pierwszyIndeks3);
	x++;

	register hipfftReal indeks4=indeksy[x];
	register int pierwszyIndeks4=floorf(indeks4);
	register int drugiIndeks4=pierwszyIndeks4+1;
	wynik[x]=sygnal[pierwszyIndeks4]*(drugiIndeks4-indeks4)+sygnal[drugiIndeks4]*(indeks4-pierwszyIndeks4);*/
}

__global__ void Padding(const hipfftComplex* wejscie, hipfftComplex* wyjscie)
{
	int x=blockDim.x*blockIdx.x+threadIdx.x;

	if(x>N/2 && x<N*2+1)
		wyjscie[x].x=wyjscie[x].y=0;
	else if(x<N/2)
	{
		wyjscie[x].x=wejscie[x].x/2;
		wyjscie[x].y=wejscie[x].y/2;
	}
	else if(x==N/2)
	{
		wyjscie[x].x=wejscie[x].x/(2*N);
		wyjscie[x].y=wejscie[x].y/(-2*N);
	}
}

void ZapiszWynikDoCsv(const hipfftReal* sygnal, const hipfftReal* wynik, const char* sciezkaPliku)
{
	FILE* plik=fopen(sciezkaPliku, "wt");

	for(int i=0; i<N; i++)
	{
		fprintf(plik, "%d\t", i);
		fprintf(plik, "%f\t", sygnal[i]);
		fprintf(plik, "%f\t\n", wynik[i]);
	}

	fclose(plik);
}

float _tmain(float argc, _TCHAR* argv[])
{
	if(InicjujCuda())
	{
		hipfftReal hX[N];
		/*hipfftComplex hY[N];
		hipfftComplex hZ[2*N+1];
		hipfftReal hWynik[4*N];*/
		hipfftReal hWynikOstateczny[N];
		hipfftReal indeksy[N*2];
		hipfftReal* dX;
		hipfftComplex* dY;
		hipfftComplex* dZ;
		hipfftReal* dWynik;
		hipfftReal* dWynikOstateczny;
		hipfftReal* dIndeksy;
		FILE* plik=fopen("E:\\cuda\\LAB7\\indeksy.bin", "rb");
		hipfftHandle plan;
		hipfftHandle plan2;

		fread(indeksy, sizeof(hipfftReal), 2*N, plik);
		fclose(plik);
		hipMalloc((void**)&dIndeksy, N*2*sizeof(hipfftReal));
		hipMalloc((void**)&dX, N*sizeof(hipfftReal));
		hipMalloc((void**)&dY, N*sizeof(hipfftComplex));
		hipMalloc((void**)&dZ, (2*N+1)*sizeof(hipfftComplex));
		hipMalloc((void**)&dWynik, 4*N*sizeof(hipfftReal));
		hipMalloc((void**)&dWynikOstateczny, N*sizeof(hipfftReal));

		for(int i=0; i<N; i++)
		{
			hX[i]=(hipfftReal)(i%modulo);
			indeksy[i]=indeksy[i*2]/2;
		}

		hipMemcpy(dX, hX, N*sizeof(hipfftReal), hipMemcpyHostToDevice);
		hipMemcpy(dIndeksy, indeksy, N*2*sizeof(hipfftReal), hipMemcpyHostToDevice);
		hipfftPlan1d(&plan, N, HIPFFT_R2C, 1);
		hipfftExecR2C(plan, dX, dY);

		Padding <<< 3, N >>> (dY, dZ);

		hipfftPlan1d(&plan2, 4*N, HIPFFT_C2R, 1);
		hipfftExecC2R(plan2, dZ, dWynik);

		Przeprobkuj4 <<< 1, N >>> (dWynik, dIndeksy, dWynikOstateczny);

		hipMemcpy(hWynikOstateczny, dWynikOstateczny, N*sizeof(hipfftReal), hipMemcpyDeviceToHost);

		for(int i=0; i<N; i++)
		{
			std::cout << hWynikOstateczny[i];

			getchar();
		}

		system("pause");
		hipFree(dIndeksy);
		hipFree(dX);
		hipFree(dY);
		hipFree(dZ);
		hipFree(dWynik);
		hipFree(dWynikOstateczny);
		hipfftDestroy(plan);
		hipfftDestroy(plan2);
		ZapiszWynikDoCsv(hX, hWynikOstateczny, "wynik.csv");
	}

	return 0;
}