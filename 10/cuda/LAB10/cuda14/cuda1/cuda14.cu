#include "hip/hip_runtime.h"
// cuda1.cpp : Defines the entry pofloat for the console application.
//

#include "stdafx.h"
#include <iostream>
#include <Windows.h>
#include <ctime>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>

static const int N=2048;
static const int modulo=20;

bool InicjujCuda()
{
	int liczbaUrzadzen;

	hipGetDeviceCount(&liczbaUrzadzen);

	if(liczbaUrzadzen>0)
	{
		if(hipSetDevice(0)==hipSuccess)
			return true;
		else
			return false;
	}
	else
		return false;
}

__global__ void Przeprobkuj4(const float* sygnal, const float* indeksy, float* wynik)
{
	register int x=(blockIdx.x*blockDim.x+threadIdx.x)*4;

	register float indeks1=indeksy[x];
	register int pierwszyIndeks1=floorf(indeks1);
	register int drugiIndeks1=pierwszyIndeks1+1;
	wynik[x]=sygnal[pierwszyIndeks1]*(drugiIndeks1-indeks1)+sygnal[drugiIndeks1]*(indeks1-pierwszyIndeks1);
	x++;

	register float indeks2=indeksy[x];
	register int pierwszyIndeks2=floorf(indeks2);
	register int drugiIndeks2=pierwszyIndeks2+1;
	wynik[x]=sygnal[pierwszyIndeks2]*(drugiIndeks2-indeks2)+sygnal[drugiIndeks2]*(indeks2-pierwszyIndeks2);
	x++;

	register float indeks3=indeksy[x];
	register int pierwszyIndeks3=floorf(indeks3);
	register int drugiIndeks3=pierwszyIndeks3+1;
	wynik[x]=sygnal[pierwszyIndeks3]*(drugiIndeks3-indeks3)+sygnal[drugiIndeks3]*(indeks3-pierwszyIndeks3);
	x++;

	register float indeks4=indeksy[x];
	register int pierwszyIndeks4=floorf(indeks4);
	register int drugiIndeks4=pierwszyIndeks4+1;
	wynik[x]=sygnal[pierwszyIndeks4]*(drugiIndeks4-indeks4)+sygnal[drugiIndeks4]*(indeks4-pierwszyIndeks4);
	x++;
}

void ZapiszWynikDoCsv(const float* sygnal, const float* wynik, const char* sciezkaPliku)
{
	FILE* plik=fopen(sciezkaPliku, "wt");

	for(int i=0; i<N; i++)
	{
		fprintf(plik, "%d\t", i);
		fprintf(plik, "%f\t", sygnal[i]);
		fprintf(plik, "%f\t\n", wynik[i]);
	}

	fclose(plik);
}

float _tmain(float argc, _TCHAR* argv[])
{
	if(InicjujCuda())
	{
		hipfftReal hX[N];
		hipfftComplex hY[N];
		float indeksy[N];
		hipfftReal* dX;
		hipfftComplex* dY;
		float* dIndeksy;
		//FILE* plik=fopen("E:\\cuda\\LAB7\\indeksy.bin", "rb");
		hipfftHandle plan;

		//fread(indeksy, sizeof(float), N, plik);
		//fclose(plik);
		hipMalloc((void**)&dX, N*sizeof(hipfftReal));
		hipMalloc((void**)&dY, N*sizeof(hipfftComplex));

		for(int i=0; i<N; i++)
			hX[i]=(float)(i%modulo);

		hipMemcpy(dX, hX, N*sizeof(hipfftReal), hipMemcpyHostToDevice);
		hipfftPlan1d(&plan, N, HIPFFT_R2C, 1);
		hipfftExecR2C(plan, dX, dY);
		hipMemcpy(hY, dY, N*sizeof(hipfftReal), hipMemcpyDeviceToHost);

		for(int i=0; i<N; i++)
		{
			std::cout << i<< "\t" << hY[i].x << "\t" << hY[i].y;
			
			getchar();
		}

		/*hipMalloc((void**)&dA, rozmiarWBajtach);
		hipMalloc((void**)&dB, rozmiarWBajtach);
		hipMalloc((void**)&dIndeksy, rozmiarWBajtach);
		hipMemcpy(dA, hA, rozmiarWBajtach, hipMemcpyHostToDevice);
		hipMemcpy(dIndeksy, indeksy, rozmiarWBajtach, hipMemcpyHostToDevice);

		Przeprobkuj4 <<< 1, N/4 >>> (dA, dIndeksy, dB);	

		hipMemcpy(hB, dB, rozmiarWBajtach, hipMemcpyDeviceToHost);*/

		system("pause");
		hipFree(dX);
		hipFree(dY);
		hipfftDestroy(plan);
		//ZapiszWynikDoCsv(hX, hB, "wynik.csv");
	}

	return 0;
}