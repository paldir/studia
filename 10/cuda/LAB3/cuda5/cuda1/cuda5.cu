#include "hip/hip_runtime.h"
// cuda1.cpp : Defines the entry point for the console application.
//

#include "stdafx.h"
#include <iostream>
#include <Windows.h>
#include <hip/hip_runtime.h>

static const int N=3;

bool InicjujCuda()
{
	int liczbaUrzadzen;

	hipGetDeviceCount(&liczbaUrzadzen);

	if(liczbaUrzadzen>0)
	{
		if(hipSetDevice(0)==hipSuccess)
			return true;
		else
			return false;
	}
	else
		return false;
}

__global__ void IloczynMacierzy(const float* dA, const float* dB, float* wynik)
{
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int x=bx*blockDim.x+tx;
	int y=by*blockDim.y+ty;
	float suma=0;

	for(int k=0; k<N; k++)
		suma+=dA[y*N+k]*dB[k*N+x];

	wynik[y*N+x]=suma;
}

int _tmain(int argc, _TCHAR* argv[])
{
	if(InicjujCuda())
	{
		const int liczbaElementow=N*N;
		float hA[liczbaElementow];
		float hB[liczbaElementow];
		float hC[liczbaElementow];
		float* dA;
		float* dB;
		float* dC;
		int rozmiarWBajtach=liczbaElementow*sizeof(float);
		dim3 bloki(N, N);

		for(int i=0; i<liczbaElementow;i++)
			hA[i]=hB[i]=i;

		hipMalloc((void**)&dA, rozmiarWBajtach);
		hipMalloc((void**)&dB, rozmiarWBajtach);
		hipMalloc((void**)&dC, rozmiarWBajtach);
		hipMemcpy(dA, hA, rozmiarWBajtach, hipMemcpyHostToDevice);
		hipMemcpy(dB, hB, rozmiarWBajtach, hipMemcpyHostToDevice);

		IloczynMacierzy <<< 1, bloki >>> (dA, dB, dC);

		hipMemcpy(hC, dC, rozmiarWBajtach, hipMemcpyDeviceToHost);

		for(int i=0; i<liczbaElementow; i++)
		{
			if(i%N==0)
				std::cout << std::endl;
			
			std::cout << hC[i] << "\t";
		}

		hipFree(dA);
		hipFree(dB);
		hipFree(dC);

		system("pause");
	}

	return 0;
}