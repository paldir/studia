#include "hip/hip_runtime.h"
// cuda1.cpp : Defines the entry point for the console application.
//

#include "stdafx.h"
#include <iostream>
#include <Windows.h>
#include <hip/hip_runtime.h>

static const int N=64;
static const int nKafelka=N/2;

static const int nWKafelkach=N/nKafelka;
static const int liczbaElementowWKafelku=nKafelka*nKafelka;

bool InicjujCuda()
{
	int liczbaUrzadzen;

	hipGetDeviceCount(&liczbaUrzadzen);

	if(liczbaUrzadzen>0)
	{
		if(hipSetDevice(0)==hipSuccess)
			return true;
		else
			return false;
	}
	else
		return false;
}

void IloczynMacierzyCpu(const float* A, const float* B, float* wynik)
{
	for(int i=0; i<N; i++)
		for(int j=0; j<N; j++)
		{
			float suma=0;

			for(int k=0; k<N; k++)
				suma+=A[j*N+k]*B[k*N+i];

			wynik[j*N+i]=suma;
		}
}

__global__ void IloczynMacierzyGpu(const float* dA, const float* dB, float* wynik)
{
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int x=bx*blockDim.x+tx;
	int y=by*blockDim.y+ty;
	float suma=0;

	for(int k=0; k<N; k++)
		suma+=dA[y*N+k]*dB[k*N+x];

	wynik[y*N+x]=suma;
}

__global__ void IloczynMacierzyGpuKafelki(const float* dA, const float* dB, float* wynik)
{
	int wiersz=blockIdx.y*nKafelka+threadIdx.y;
	int kolumna=blockIdx.x*nKafelka+threadIdx.x;
	float suma=0;

	for(int k=0; k<N; k++)
		suma+=dA[wiersz*N+k]*dB[k*N+kolumna];

	wynik[wiersz*N+kolumna]=suma;
}

__global__ void IloczynMacierzyGpuKafelkiTurbo(const float* dA, const float* dB, float* wynik)
{
	__shared__ float kafelekA[liczbaElementowWKafelku];
	__shared__ float kafelekB[liczbaElementowWKafelku];
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int wiersz=blockIdx.y*nKafelka+ty;
	int kolumna=blockIdx.x*nKafelka+tx;
	float suma=0;
	int indeksKafelka=ty*nKafelka+tx;
	int indeksGlobalny=wiersz*N+kolumna;
	kafelekA[indeksKafelka]=dA[indeksGlobalny];
	kafelekB[indeksKafelka]=dB[indeksGlobalny];

	__syncthreads();

	for(int k=0; k<nKafelka; k++)
		suma+=kafelekA[ty*nKafelka+k]*kafelekB[k*nKafelka+tx];

	wynik[indeksGlobalny]+=suma;
}

void Wyswietl(const float* macierz, const int liczbaElementow)
{
	for(int i=0; i<liczbaElementow; i++)
	{
		if(i%N==0)
			std::cout << std::endl;

		std::cout << macierz[i] << " ";
	}
}

int _tmain(int argc, _TCHAR* argv[])
{
	if(InicjujCuda())
	{
		const int liczbaElementow=N*N;
		float hA[liczbaElementow];
		float hB[liczbaElementow];
		float hC[liczbaElementow];
		float* dA;
		float* dB;
		float* dC;
		int rozmiarWBajtach=liczbaElementow*sizeof(float);
		dim3 siatka(nWKafelkach, nWKafelkach);
		dim3 bloki(nKafelka, nKafelka);

		for(int i=0; i<liczbaElementow;i++)
		{
			hA[i]=1.0f;
			hB[i]=1.0f;
		}

		hipMalloc((void**)&dA, rozmiarWBajtach);
		hipMalloc((void**)&dB, rozmiarWBajtach);
		hipMalloc((void**)&dC, rozmiarWBajtach);
		hipMemcpy(dA, hA, rozmiarWBajtach, hipMemcpyHostToDevice);
		hipMemcpy(dB, hB, rozmiarWBajtach, hipMemcpyHostToDevice);

		//IloczynMacierzyCpu(hA, hB, hC);
		IloczynMacierzyGpuKafelkiTurbo <<< siatka, bloki >>> (dA, dB, dC);

		hipMemcpy(hC, dC, rozmiarWBajtach, hipMemcpyDeviceToHost);

		Wyswietl(hA, liczbaElementow);
		std::cout << std::endl;
		Wyswietl(hB, liczbaElementow);
		std::cout << std::endl;
		Wyswietl(hC, liczbaElementow);
		std::cout << std::endl;

		hipFree(dA);
		hipFree(dB);
		hipFree(dC);

		system("pause");
	}

	return 0;
}