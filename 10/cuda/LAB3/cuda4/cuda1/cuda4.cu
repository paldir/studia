#include "hip/hip_runtime.h"
// cuda1.cpp : Defines the entry point for the console application.
//

#include "stdafx.h"
#include <iostream>
#include <Windows.h>
#include <hip/hip_runtime.h>

static const int liczbaBlokow=1024;

bool InicjujCuda()
{
	int liczbaUrzadzen;

	hipGetDeviceCount(&liczbaUrzadzen);

	if(liczbaUrzadzen>0)
	{
		if(hipSetDevice(0)==hipSuccess)
			return true;
		else
			return false;
	}
	else
		return false;
}

__global__ void IloczynSkalarny2(const float* dA, const float* dB, float* wynik)
{
	__shared__ float iloczyny[liczbaBlokow];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	int x=bx*blockDim.x+tx;
	iloczyny[tx]=dA[x]*dB[x];

	__syncthreads();

	if(tx==0)
	{
		float suma=0;

		for(int i=0; i<blockDim.x; i++)
			suma+=iloczyny[i];

		wynik[bx]=suma;
	}
}

__global__ void IloczynSkalarny3(const float* dA, const float* dB, float* wynik)
{
	__shared__ float iloczyny[liczbaBlokow];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	int x=bx*blockDim.x+tx;
	iloczyny[tx]=dA[x]*dB[x];

	for(int i=2; i<=liczbaBlokow; i*=2)
	{
		__syncthreads();

		if(tx%i==0)
			iloczyny[tx]+=iloczyny[tx+i/2];
	}

	if(tx==0)
		wynik[bx]=iloczyny[0];
}

__global__ void IloczynSkalarny4(const float* dA, const float* dB, float* wynik)
{
	__shared__ float iloczyny[liczbaBlokow];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	int x=bx*blockDim.x+tx;
	iloczyny[tx]=dA[x]*dB[x];

	for(int i=liczbaBlokow >> 1; i>0; i>>=1)
	{
		__syncthreads();

		if(tx<i)
			iloczyny[tx]+=iloczyny[tx+i];
	}

	if(tx==0)
		wynik[bx]=iloczyny[0];
}

int _tmain(int argc, _TCHAR* argv[])
{
	if(InicjujCuda())
	{
		const int liczbaWszystkichWatkow=liczbaBlokow*liczbaBlokow;
		float* hA=new float[liczbaWszystkichWatkow];
		float* hB=new float[liczbaWszystkichWatkow];
		float hWynik[liczbaBlokow];
		float* dA;
		float* dB;
		float* dWynik;
		int rozmiarWektoraWBajtach=liczbaWszystkichWatkow*sizeof(float);
		int rozmiarWynikuWBajtach=liczbaBlokow*sizeof(float);
		LARGE_INTEGER tyknieciaNaSekunde, tic, toc;

		for(int i=0; i<liczbaWszystkichWatkow;i++)
		{
			hA[i]=hB[i]=1.0f;
		}

		QueryPerformanceFrequency(&tyknieciaNaSekunde);

		hipMalloc((void**)&dA, rozmiarWektoraWBajtach);
		hipMalloc((void**)&dB, rozmiarWektoraWBajtach);
		hipMalloc((void**)&dWynik, rozmiarWynikuWBajtach);
		hipMemcpy(dA, hA, rozmiarWektoraWBajtach, hipMemcpyHostToDevice);
		hipMemcpy(dB, hB, rozmiarWektoraWBajtach, hipMemcpyHostToDevice);

		//------------------------------------------------------------------------------

		IloczynSkalarny2 <<< liczbaBlokow, liczbaBlokow >>> (dA, dB, dWynik);

		hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);

		IloczynSkalarny2 <<< liczbaBlokow, liczbaBlokow >>> (dA, dB, dWynik);

		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);

		std::cout << "2: " << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;

		//------------------------------------------------------------------------------

		IloczynSkalarny3 <<< liczbaBlokow, liczbaBlokow >>> (dA, dB, dWynik);

		hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);

		IloczynSkalarny3 <<< liczbaBlokow, liczbaBlokow >>> (dA, dB, dWynik);

		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);

		std::cout << "3: " << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;

		//------------------------------------------------------------------------------

		IloczynSkalarny4 <<< liczbaBlokow, liczbaBlokow >>> (dA, dB, dWynik);

		hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);

		IloczynSkalarny4 <<< liczbaBlokow, liczbaBlokow >>> (dA, dB, dWynik);

		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);

		std::cout << "4: " << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;

		//------------------------------------------------------------------------------

		/*hipMemcpy(hWynik, dWynik, rozmiarWynikuWBajtach, hipMemcpyDeviceToHost);

		for(int i=0; i<liczbaBlokow;i++)
		std::cout << hWynik[i] << std::endl;*/

		system("pause");

		delete[] hA;
		delete[] hB;
		hipFree(dA);
		hipFree(dB);
		hipFree(dWynik);
	}

	return 0;
}