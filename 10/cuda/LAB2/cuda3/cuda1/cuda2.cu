#include "hip/hip_runtime.h"
// cuda1.cpp : Defines the entry point for the console application.
//

#include "stdafx.h"
#include <iostream>
#include <hip/hip_runtime.h>

#define rozmiarWektora 1024

bool InicjujCuda()
{
	int liczbaUrzadzen;

	hipGetDeviceCount(&liczbaUrzadzen);

	if(liczbaUrzadzen>0)
	{
		if(hipSetDevice(0)==hipSuccess)
			return true;
		else
			return false;
	}
	else
		return false;
}

__global__ void IloczynSkalarny2(const float* dA, const float* dB, float* wynik)
{
	__shared__ float iloczyny[rozmiarWektora];
	int x=threadIdx.x;
	iloczyny[x]=dA[x]*dB[x];

	__syncthreads();

	if(x==0)
	{
		float suma=0;

		for(int i=0; i<blockDim.x; i++)
			suma+=iloczyny[i];

		wynik[0]=suma;
	}
}

__global__ void IloczynSkalarny3(const float* dA, const float* dB, float* wynik)
{
	__shared__ float iloczyny[rozmiarWektora];
	int x=threadIdx.x;
	iloczyny[x]=dA[x]*dB[x];

	for(int i=2; i<=rozmiarWektora; i*=2)
	{
		__syncthreads();

		if(x%i==0)
			iloczyny[x]+=iloczyny[x+i/2];
	}

	if(x==0)
		wynik[0]=iloczyny[0];
}

__global__ void IloczynSkalarny4(const float* dA, const float* dB, float* wynik)
{
	__shared__ float iloczyny[rozmiarWektora];
	int x=threadIdx.x;
	iloczyny[x]=dA[x]*dB[x];

	for(int i=2; i<=rozmiarWektora; i*=2)
	{
		__syncthreads();

		if(x%i==0)
		{
			int nowyX=x/i;

			iloczyny[nowyX]+=iloczyny[nowyX+1];
		}
	}

	if(x==0)
		wynik[0]=iloczyny[0];
}

int _tmain(int argc, _TCHAR* argv[])
{
	if(InicjujCuda())
	{
		float hA[rozmiarWektora];
		float hB[rozmiarWektora];
		float hWynik;
		float* dA;
		float* dB;
		float* dWynik;
		int rozmiarWektoraWBajtach=rozmiarWektora*sizeof(float);

		for(int i=0; i<rozmiarWektora;i++)
		{
			hA[i]=hB[i]=1.0f;
		}

		for(int i=0; i<rozmiarWektora; i++)
		{
			std::cout << hA[i] << "\t" << hB[i] << std::endl;
		}

		hipMalloc((void**)&dA, rozmiarWektoraWBajtach);
		hipMalloc((void**)&dB, rozmiarWektoraWBajtach);
		hipMalloc((void**)&dWynik, sizeof(float));
		hipMemcpy(dA, hA, rozmiarWektoraWBajtach, hipMemcpyHostToDevice);
		hipMemcpy(dB, hB, rozmiarWektoraWBajtach, hipMemcpyHostToDevice);

		//IloczynSkalarny2 <<< 1, rozmiarWektorow >>> (dA, dB, dWynik);
		//IloczynSkalarny3 <<< 1, rozmiarWektora >>> (dA, dB, dWynik);
		IloczynSkalarny4 <<< 1, rozmiarWektora >>> (dA, dB, dWynik);

		hipMemcpy(&hWynik, dWynik, sizeof(float), hipMemcpyDeviceToHost);

		std::cout << hWynik << std::endl;

		system("pause");

		hipFree(dA);
		hipFree(dB);
		hipFree(dWynik);
	}

	return 0;
}