#include "hip/hip_runtime.h"
// cuda1.cpp : Defines the entry point for the console application.
//

#include "stdafx.h"
#include <iostream>
#include <Windows.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

static const int N=64;
static const int nKafelka=N/2;

static const int nWKafelkach=N/nKafelka;
//static const int liczbaElementowWKafelku=nKafelka*nKafelka;

bool InicjujCuda()
{
	int liczbaUrzadzen;

	hipGetDeviceCount(&liczbaUrzadzen);

	if(liczbaUrzadzen>0)
	{
		if(hipSetDevice(0)==hipSuccess)
			return true;
		else
			return false;
	}
	else
		return false;
}

__global__ void IloczynMacierzyGpuKafelkiTurbo(const float* dA, const float* dB, float* wynik)
{
	__shared__ float kafelekA[1024];
	__shared__ float kafelekB[1024];
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int wiersz=blockIdx.y*nKafelka+ty;
	int kolumna=blockIdx.x*nKafelka+tx;
	float suma=0;

	for(int i=0; i<nWKafelkach; i++)
	{
		int indeksKafelka=ty*nKafelka+tx;
		kafelekA[indeksKafelka]=dA[wiersz*N+i*nKafelka+tx];
		kafelekB[indeksKafelka]=dB[(i*nKafelka+ty)*N+kolumna];

		__syncthreads();

		for(int k=0; k<nKafelka; k++)
			suma+=kafelekA[ty*nKafelka+k]*kafelekB[k*nKafelka+tx];

		__syncthreads();
	}

	wynik[wiersz*N+kolumna]=suma;
}

void Wyswietl(const float* macierz, const int liczbaElementow)
{
	for(int i=0; i<liczbaElementow; i++)
	{
		if(i%N==0)
			std::cout << std::endl;

		std::cout << macierz[i] << " ";
	}
}

int _tmain(int argc, _TCHAR* argv[])
{
	if(InicjujCuda())
	{
		const int liczbaElementow=N*N;
		float hA[liczbaElementow];
		float hB[liczbaElementow];
		float hC[liczbaElementow];
		float hC2[liczbaElementow];
		float* dA;
		float* dB;
		float* dC;
		float* dC2;
		int rozmiarWBajtach=liczbaElementow*sizeof(float);
		dim3 siatka(nWKafelkach, nWKafelkach);
		dim3 bloki(nKafelka, nKafelka);
		hipblasHandle_t uchwyt;
		float alfa=1;
		float beta=0;
		LARGE_INTEGER tyknieciaNaSekunde, tic, toc;

		hipblasCreate(&uchwyt);
		QueryPerformanceFrequency(&tyknieciaNaSekunde);

		for(int i=0; i<liczbaElementow;i++)
		{
			hA[i]=1.0f;
			hB[i]=1.0f;
		}

		hipMalloc((void**)&dA, rozmiarWBajtach);
		hipMalloc((void**)&dB, rozmiarWBajtach);
		hipMalloc((void**)&dC, rozmiarWBajtach);
		hipMalloc((void**)&dC2, rozmiarWBajtach);
		hipMemcpy(dA, hA, rozmiarWBajtach, hipMemcpyHostToDevice);
		hipMemcpy(dB, hB, rozmiarWBajtach, hipMemcpyHostToDevice);

		/*Wyswietl(hA, liczbaElementow);
		std::cout << std::endl;
		Wyswietl(hB, liczbaElementow);
		std::cout << std::endl;*/

		IloczynMacierzyGpuKafelkiTurbo <<< siatka, bloki >>> (dA, dB, dC);
		hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);
		IloczynMacierzyGpuKafelkiTurbo <<< siatka, bloki >>> (dA, dB, dC);
		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);
		hipMemcpy(hC, dC, rozmiarWBajtach, hipMemcpyDeviceToHost);
		//Wyswietl(hC, liczbaElementow);

		std::cout << "1: " << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;

		hipblasSgemm(uchwyt, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alfa, dA, N, dB, N, &beta, dC2, N);
		hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);
		hipblasSgemm(uchwyt, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alfa, dA, N, dB, N, &beta, dC2, N);
		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);
		hipMemcpy(hC2, dC2, rozmiarWBajtach, hipMemcpyDeviceToHost);
		//Wyswietl(hC2, liczbaElementow);

		std::cout << "2: " << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;
		std::cout << std::endl;

		hipFree(dA);
		hipFree(dB);
		hipFree(dC);
		hipblasDestroy(uchwyt);

		system("pause");
	}

	return 0;
}