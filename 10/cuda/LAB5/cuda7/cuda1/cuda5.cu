#include "hip/hip_runtime.h"
// cuda1.cpp : Defines the entry point for the console application.
//

#include "stdafx.h"
#include <iostream>
#include <Windows.h>
#include <ctime>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipblas.h>

static const int dlugoscWektora=1500;
static const int rozmiarOkna=5;

static const int polowaRozmiaruOkna=rozmiarOkna/2;

bool InicjujCuda()
{
	int liczbaUrzadzen;

	hipGetDeviceCount(&liczbaUrzadzen);

	if(liczbaUrzadzen>0)
	{
		if(hipSetDevice(0)==hipSuccess)
			return true;
		else
			return false;
	}
	else
		return false;
}

__device__ __host__ void SortowanieOkna(int* tablica)
{
	bool zamiana;

	do
	{
		zamiana=false;

		for(int i=1; i<rozmiarOkna; i++)
			if(tablica[i-1]>tablica[i])
			{
				int tmp=tablica[i-1];
				tablica[i-1]=tablica[i];
				tablica[i]=tmp;
				zamiana=true;
			}
	}
	while(zamiana);
}

__global__ void Filtrowanie(const int* dA, int* wynik)
{
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	int x=bx*blockDim.x+tx;
	int okno[rozmiarOkna];

	if(x>=polowaRozmiaruOkna && x<dlugoscWektora-polowaRozmiaruOkna)
	{
		for(int i=0; i<rozmiarOkna; i++)
			okno[i]=dA[x-polowaRozmiaruOkna+i];

		SortowanieOkna(okno);

		wynik[x]=okno[polowaRozmiaruOkna];
	}
}

int _tmain(int argc, _TCHAR* argv[])
{
	const int maksimum=100;
	const int dlugoscLiczb=5;

	if(InicjujCuda())
	{
		int hA[dlugoscWektora];
		int hB[dlugoscWektora];
		int* dA;
		int* dB;
		int rozmiarWBajtach=dlugoscWektora*sizeof(int);

		srand((unsigned int)time(NULL));

		for(int i=0; i<dlugoscWektora;i++)
		{
			int liczba=std::rand()%maksimum;
			hA[i]=liczba;

			std::cout << liczba << " ";
		}

		std::cout << std::endl;

		hipMalloc((void**)&dA, rozmiarWBajtach);
		hipMalloc((void**)&dB, rozmiarWBajtach);
		hipMemcpy(dA, hA, rozmiarWBajtach, hipMemcpyHostToDevice);

		Filtrowanie <<< dlugoscWektora/1024+1, 1024 >>> (dA, dB);

		hipMemcpy(hB, dB, rozmiarWBajtach, hipMemcpyDeviceToHost);

		std::cout << std::endl;

		for(int x=polowaRozmiaruOkna; x<dlugoscWektora-polowaRozmiaruOkna; x++)
		{
			int okno[rozmiarOkna];

			for(int i=0; i<rozmiarOkna; i++)
			{
				okno[i]=hA[x-polowaRozmiaruOkna+i];

				std::cout << std::setw(dlugoscLiczb) << okno[i];
			}

			SortowanieOkna(okno);

			std::cout << " -> ";

			for(int i=0; i<rozmiarOkna; i++)
				std::cout << std::setw(dlugoscLiczb) << okno[i];

			std::cout << std::endl;
		}

		std::cout << std::endl;

		for(int i=0; i<dlugoscWektora; i++)
			std::cout << hB[i] << " ";

		std::cout << std::endl;

		hipFree(dA);
		hipFree(dB);

		system("pause");
	}

	return 0;
}