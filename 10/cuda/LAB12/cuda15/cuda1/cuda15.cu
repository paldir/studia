#include "hip/hip_runtime.h"
// cuda1.cpp : Defines the entry pofloat for the console application.
//

#include "stdafx.h"
#include <iostream>
#include <Windows.h>
#include <ctime>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>

static const int N=128;
static const int modulo=32;
static const int liczbaElementow=N*N;

__constant__ float macierzStala[liczbaElementow];
texture<float> tekstury;

bool InicjujCuda()
{
	int liczbaUrzadzen;

	hipGetDeviceCount(&liczbaUrzadzen);

	if(liczbaUrzadzen>0)
	{
		if(hipSetDevice(0)==hipSuccess)
			return true;
		else
			return false;
	}
	else
		return false;
}

__global__ void Spojny(const float* macierz1, float* macierz2)
{
	int x=blockDim.x*blockIdx.x+threadIdx.x;

	macierz2[x]=macierz1[x]+32;
}

__global__ void Niespojny(const float* macierz1, float* macierz2)
{
	int x=blockDim.x*threadIdx.x+blockIdx.x;

	macierz2[x]=macierz1[x]+32;
}

__global__ void PamiecStala(float* macierz2)
{
	int x=blockDim.x*blockIdx.x+threadIdx.x;

	macierz2[x]=macierzStala[x]+32;
}

__global__ void JedenBlokUrzadzenie(const float* macierz1, float* macierz2)
{
	int zrodlo=blockIdx.x;
	int cel=blockDim.x*zrodlo+threadIdx.x;

	macierz2[cel]=macierz1[zrodlo]+32;
}

__global__ void JedenBlokStala(float* macierz2)
{
	int zrodlo=blockIdx.x;
	int cel=blockDim.x*zrodlo+threadIdx.x;

	macierz2[cel]=macierzStala[zrodlo]+32;
}

__global__ void Tekstury(float* macierz2)
{
	int x=blockDim.x*blockIdx.x+threadIdx.x;

	macierz2[x]=tex1Dfetch(tekstury, x)+32;
}

float _tmain(float argc, _TCHAR* argv[])
{
	if(InicjujCuda())
	{
		float* macierzH=new float[liczbaElementow];
		float* macierzZrodlowaD;
		float* macierzDocelowaD;
		float* macierzZrodlowaTekstur;
		size_t rozmiarMacierzy=sizeof(float)*liczbaElementow;
		LARGE_INTEGER tyknieciaNaSekunde, tic, toc;

		for(int i=0; i<liczbaElementow; i++)
			macierzH[i]=(float)(i%modulo);

		hipHostAlloc((void**)&macierzZrodlowaD, rozmiarMacierzy, hipHostMallocDefault);
		hipHostAlloc((void**)&macierzDocelowaD, rozmiarMacierzy, hipHostMallocDefault);
		hipHostAlloc((void**)&macierzZrodlowaTekstur, rozmiarMacierzy, hipHostMallocDefault);
		hipMemcpy(macierzZrodlowaD, macierzH, rozmiarMacierzy, hipMemcpyHostToDevice);
		hipMemcpy(macierzZrodlowaTekstur, macierzH, rozmiarMacierzy, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(macierzStala), macierzH, rozmiarMacierzy); 
		hipBindTexture(0, tekstury, macierzZrodlowaTekstur, rozmiarMacierzy);
		QueryPerformanceFrequency(&tyknieciaNaSekunde);

		hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);

		Niespojny <<< N, N >>> (macierzZrodlowaD, macierzDocelowaD);

		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);

		std::cout << "Dostep niespojny:\t\t" << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;


		hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);

		Spojny <<< N, N >>> (macierzZrodlowaD, macierzDocelowaD);

		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);

		std::cout << "Dostep spojny:\t\t\t" << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;


		hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);

		PamiecStala <<< N, N >>> (macierzDocelowaD);

		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);

		std::cout << "Pamiec stala:\t\t\t" << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;


		/*hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);

		JedenBlokUrzadzenie <<< N, N >>> (macierzZrodlowaD, macierzDocelowaD);

		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);

		std::cout << "Jeden blok - urzadzenie:\t" << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;


		hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);

		JedenBlokStala <<< N, N >>> (macierzDocelowaD);

		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);

		std::cout << "Jeden blok - pamiec stala:\t" << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;*/


		hipDeviceSynchronize();
		QueryPerformanceCounter(&tic);

		Tekstury <<< N, N >>> (macierzDocelowaD);

		hipDeviceSynchronize();
		QueryPerformanceCounter(&toc);

		std::cout << "Tekstury:\t\t\t" << (double)(toc.QuadPart-tic.QuadPart)/tyknieciaNaSekunde.QuadPart*1000 << std::endl;

		delete[] macierzH;
		hipFree(macierzZrodlowaD);
		hipFree(macierzDocelowaD);
		hipUnbindTexture(tekstury);

		std::cout << std::endl;

		system("pause");
	}

	return 0;
}